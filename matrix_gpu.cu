#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>

#define MATRIX_SIZE 1024/*行列１辺の数*/
#define BLOCK_SIZE 16

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int main(int argc, char** argv){
  unsigned int matrixSize = sizeof(unsigned int) * MATRIX_SIZE * MATRIX_SIZE;

  int* hMatrixA;
  int* hMatrixB;
  int* hMatrixC;
  hMatrixA = (int*)malloc(matrixSize);
  hMatrixB = (int*)malloc(matrixSize);

  /*初期値設定*/
  unsigned int col_idx, row_idx;
  for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++){
    for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++){
      hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024*1024);
      hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024*1024);
    }
  }

  /*デバイス側の変数設定*/
  int* dMatrixA;
  int* dMatrixB;
  int* dMatrixC;

  /*デバイスメモリ領域の確保*/
  checkCudaErrors(hipMalloc((void**)&dMatrixA, matrixSize));
  checkCudaErrors(hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void**)&dMatrixB, matrixSize));
  checkCudaErrors(hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void**)&dMatrixC, matrixSize));

  /*ブロックサイズとグリッドサイズの設定*/
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(MATRIX_SIZE/BLOCK_SIZE, MATRIX_SIZE/BLOCK_SIZE);

  /*タイマーを作成して計測開始*/
  cudaevent_t start;
  hipEvent_t stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipEventRecord(start, NULL)); // スタート

  /*カーネルの起動*/
  matrixMul<<<grid, block>>>(dMatrixA, dMatrixB, dMatrixC);
  hipDeviceSynchronize();

  /*結果の領域確保とデバイス側からのメモリ転送*/
  hMatrixC = (int*)malloc(matrixSize);
  checkCudaErrors(hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost));

  /*タイマーを停止しかかった時間を表示*/

  checkCudaErrors(hipEventRecord(stop, NULL));
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  printf("Processing time: %f (msec)\n", msecTotal);

  /*ホスト・デバイスメモリの開放*/
  free(hMatrixA);
  free(hMatrixB);
  free(hMatrixC);
  checkCudaErrors(hipFree(dMatrixA));
  checkCudaErrors(hipFree(dMatrixB));
  checkCudaErrors(hipFree(dMatrixC));

  /*終了処理*/
  hipDeviceReset();
  exit(1);
}

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC){
  unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int scan_idx;
  unsigned int target = 0;

  /*行列の演算を行う*/
  for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
    target +=inMatrixA[col_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
    __syncthreads();
  }
  inMatrixC[col_idx * MATRIX_SIZE + row_idx] = target;
}
